#include "hip/hip_runtime.h"
// example1.cpp : Defines the entry point for the console application.
//

//#include <stdafx.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <float.h>
#include <math.h>

#include "cuPrintf.h"

void findMinMax (double x, double y);

//Maximum & Minimum variables
float minX=0.0, minY=0.0, maxX=0.0, maxY=0.0;

__device__ int Doolittle_LU_Decomposition_with_Pivoting(double *A, int pivot[], int n)
{
	int i, j, k;
	double *p_k, *p_row, *p_col;
	double max;

	//cuPrintf("nilai n %d\n",n);
	int id=0;
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			//cuPrintf("%f; ",*(A+id));
			id++;
		}
		//cuPrintf("\n");
	}

	//         For each row and column, k = 0, ..., n-1,

	for (k = 0, p_k = A; k < n; p_k += n, k++) {

		//            find the pivot row

		pivot[k] = k;
		//cuPrintf("*(p_k+k) = %f\n",*(p_k+k));
		max = fabs( *(p_k + k) );
		for (j = k + 1, p_row = p_k + n; j < n; j++, p_row += n) {
			if ( max < fabs(*(p_row + k) )){
				max = fabs(*(p_row + k));//*(p_row + k) =  *A
				pivot[k] = j;
				p_col = p_row;
			}
		}

		//     and if the pivot row differs from the current row, then
		//     interchange the two rows.

		if (pivot[k] != k)
			for (j = 0; j < n; j++) {
				max = *(p_k + j);
				*(p_k + j) = *(p_col + j);
				*(p_col + j) = max;
			}

		//                and if the matrix is singular, return error


		if ( *(p_k + k) == 0.0 ) return -1;

		//      otherwise find the lower triangular matrix elements for column k.

		for (i = k+1, p_row = p_k + n; i < n; p_row += n, i++) {
			*(p_row + k) /= *(p_k + k);
		}

		//            update remaining matrix

		for (i = k+1, p_row = p_k + n; i < n; p_row += n, i++)
			for (j = k+1; j < n; j++)
				*(p_row + j) -= *(p_row + k) * *(p_k + j);

	}

	return 0;
}


__device__ int Doolittle_LU_with_Pivoting_Solve(double *A, double B[], int pivot[],
		double x[], int n)
{
	int i, k;
	double *p_k;
	double dum;

	//         Solve the linear equation Lx = B for x, where L is a lower
	//         triangular matrix with an implied 1 along the diagonal.

	for (k = 0, p_k = A; k < n; p_k += n, k++) {
		if (pivot[k] != k) {dum = B[k]; B[k] = B[pivot[k]]; B[pivot[k]] = dum; }
		x[k] = B[k];
		for (i = 0; i < k; i++) x[k] -= x[i] * *(p_k + i);
	}

	//         Solve the linear equation Ux = y, where y is the solution
	//         obtained above of Lx = B and U is an upper triangular matrix.

	for (k = n-1, p_k = A + n*(n-1); k >= 0; k--, p_k -= n) {
		if (pivot[k] != k) {dum = B[k]; B[k] = B[pivot[k]]; B[pivot[k]] = dum; }
		for (i = k + 1; i < n; i++) x[k] -= x[i] * *(p_k + i);
		if (*(p_k + k) == 0.0) return -1;
		x[k] /= *(p_k + k);
	}

	return 0;
}

__device__ double sphericalModel (double range, double sill, double h){
	return sill*powf( (3*h/(2*range)) - 1/2*(h/range) , 3);
	//	return 10*(1-exp(-h/3.33));
}

__device__ double leastsquaredSumFunc (double *semivar, double* b0, double* h, int nrbins){
	double sum=0.0;
	int i;
	/*for (i=0 ; i<nrbins ; i++){
		printf("semivar: %lf, distBins: %lf\n", semivar[i], h[i]);
	}
	printf("b: %lf,%lf,  nrbins: %d\n", b0[0], b0[1], nrbins);*/
	for (i=0 ; i<nrbins ; i++){
		if (h[i] < b0[0]){
			sum += powf( ( b0[1]*(3*h[i]/(2*b0[0])- 1/2*(h[i]/b0[0]))*(3*h[i]/(2*b0[0])- 1/2*(h[i]/b0[0]))*(3*h[i]/(2*b0[0])- 1/2*(h[i]/b0[0]))-semivar[i]) ,2);
			//sum+=pow(sphericalModel(b0[0], b0[1], h[i])-semivar[i], 2);
		}else{
			sum+=pow(b0[1]-semivar[i], 2);
		}

	}
	return sum;
}

__global__ void on_core_process(double *dev_debugVar,int GPUIndexStart, double* ptr_xgridpoint, double* ptr_ygridpoint,double* ptrpredictionResult,
		double *x_device, double *y_device, double *z_device, double minimumX, double minimumY, int numInputData, double gridsizes,
		int GPUIndexEnd, int dimGridX_size)
{
	/**
	  Algorithms:
	  1) Specify anchor coordinates.
	  2) We calculate distance from this point.
	  3) This point will be different from each GPU thread
	  4) Closest node are constant based on radius meters (not based on square of grid)

	 */

	/**
	 * GPU THREAD ID
	 *  GRIDX_RANGE+GRIDY_RANGE                                ...
	 *   .
	 *   .
	 *   .
	 *  GRIDX_RANGE+1 GRIDX_RANGE+2                            ...      2*1000
	 *   0             1              2              3         ...       1000
	 */

	int dimBlocksize = dimGridX_size;
	int gridID = blockIdx.x+(blockIdx.y*gridDim.x);
	//retrieve thread ID
	int threadID = ((threadIdx.y)*blockDim.x+(threadIdx.x))+gridID*gridDim.x*gridDim.y;
	int dumpsize = threadID;
	threadID = threadID+GPUIndexStart;

	//we represent dimBlocksize as X axis length
	//double y_node = (double) (1*floor((float)threadID/1000));
	double y_node = (double) (1*floor((float)threadID/dimBlocksize));

	//shift load jobs into several devices
	double x_node = (double) (threadID%dimBlocksize);

	//do each thread process here
	double weightsum;
	int idx_onRange;
	double predict=0;

	if ((threadID<GPUIndexEnd)&&(threadID>=GPUIndexStart)) {
		//Calculate distance from each LIDAR input and save it into index if the distance still on the range
		// in here we assume that we have only maximum 100 closest nodes
		float x_closestNodesIndex[100];
		float y_closestNodesIndex[100];
		float z_closestNodesIndex[100];

		int gridRadius = 3;
		idx_onRange=0;
		for(int i=0;i<numInputData;i++){ //TODO to big numInputData make this program crash
			//save the value if distance is still on range
			if(( abs((x_device[i]-(x_node+minimumX)))<gridRadius)&&(abs((y_device[i]-(y_node+minimumY)))<gridRadius)){
				x_closestNodesIndex[idx_onRange] = (float)x_device[i]; //change this with pointer to enhance performance
				y_closestNodesIndex[idx_onRange] = (float)y_device[i]; //change this with pointer to enhance performance
				z_closestNodesIndex[idx_onRange] = (float)z_device[i];
				if(idx_onRange<100){ //TODO solve this function! why more than 100 is not allowed? stack problem?
					idx_onRange++;
				}
			}
		}

		//find min & max value from closestNodesIndex variable (both X & Y)
		double min_x=999999;
		double max_x=0;
		double min_y=9999999;
		double max_y=0;
		for(int i=0;i<idx_onRange;i++){
			if(x_closestNodesIndex[i]<min_x)
				min_x = x_closestNodesIndex[i];
			if(y_closestNodesIndex[i]<min_y)
				min_y = y_closestNodesIndex[i];
			if(y_closestNodesIndex[i]>max_y)
				max_y = y_closestNodesIndex[i];
			if(x_closestNodesIndex[i]>max_x)
				max_x = x_closestNodesIndex[i];
			i++;
		}

		//Calculate Semivariogram
		int nrbins = 50; //this is a parameter
		int rows = idx_onRange;
		double occIdxdistBins[50]; //parameter : nrbins
		memset(occIdxdistBins,0,sizeof(occIdxdistBins));
		double sumSqurZ[50]; //parameter : nrbins
		memset(sumSqurZ,0,sizeof(sumSqurZ));
		double distance[50]; //parameter : nrbins
		memset(distance,0,sizeof(distance));
		float ptopDistance[100][100];
		memset(ptopDistance,0,sizeof(ptopDistance));

		float predist = sqrtf((float)(powf(max_x-min_x,2)+powf(max_y-min_y,2)));
		float maxdist = predist/2;
		float delta = maxdist/2;

		for(int i=0;i<rows;i++){
			for(int j=i+1; j<rows;j++){
				ptopDistance[i][j]= sqrt((float)(powf((float)(x_closestNodesIndex[i]-x_closestNodesIndex[j]),2.0)
						+powf((float)(y_closestNodesIndex[i]-y_closestNodesIndex[j]),2.0)));

				if(ptopDistance[i][j]<maxdist){
					int idx = (int) floorf(ptopDistance[i][j]/delta);
					occIdxdistBins[idx] = occIdxdistBins[idx]+1;
					double squrZ =powf(z_closestNodesIndex[i]-z_closestNodesIndex[j],2);
					sumSqurZ[idx]=sumSqurZ[idx]+squrZ;
				}
			}
		}

		for(int i=0;i<nrbins; i++){
			if(occIdxdistBins[i] == 0.0){
				sumSqurZ[i] = 0.0;
			}else{
				sumSqurZ[i] = sumSqurZ[i]/(2*occIdxdistBins[i]);
			}
		}

		distance[0]=delta/2;
		for(int i=0; i<nrbins;i++){
			distance[i]=distance[i-1]+delta;
		}

		//SEmivariogram process is completed here


		//Starting Fitting

		double maxVario, minLs[3] = {DBL_MAX,0,0}, tmpLS[3]; // For fit
		double lagbin, semibin;
		double b0[3] = {0};

		//find maximum variogram value
		double max = 0.0;
		for(int i=0;i<nrbins;i++){
			if(sumSqurZ[i]>max)
				max = sumSqurZ[i];
		}
		maxVario = max;

		int bins = 10;
		//b0[0] = range
		//b0[1] = sill
		b0[0] = distance[nrbins-1] * 2 /3;
		b0[1] = maxVario;
		lagbin = b0[0] / bins;
		semibin = b0[1]/ bins;
		int i,j;
		for (i=0 ; i<bins ; i++){
			for (j=0 ; j<bins ; j++){
				b0[0] -= lagbin;
				tmpLS[0] = leastsquaredSumFunc(sumSqurZ, b0, distance, nrbins);
				tmpLS[1]= b0[0];
				tmpLS[2]= b0[1];
				if(minLs[0] > tmpLS[0]) {
					minLs[0] = tmpLS[0];
					minLs[1] = b0[0];
					minLs[2] = b0[1];
				}

			}
			b0 [0] = distance[nrbins-1] * 2 / 3;
			b0 [1] -= semibin;
		}

		if (minLs[1] < 0.0001){
			minLs[1] = distance[0];
		}

		double range = minLs[1];
		double sill = minLs[2];
		//		range = 200;//Hard code
		//		sill = 150;

		int N = idx_onRange+1; // we need one more column  & Row to fill 0 and 1 values (Ordinary Kriging)

		//TODO calculate distance anchor node with
		int counter_idxClosestRange =0;
		while (counter_idxClosestRange < idx_onRange){
			ptopDistance[counter_idxClosestRange][N-1] = sqrt((float)(powf((float)(x_closestNodesIndex[counter_idxClosestRange]-(x_node+minimumX)),2.0)
					+powf((float)(y_closestNodesIndex[counter_idxClosestRange]-(y_node+minimumY)),2.0)));
			counter_idxClosestRange++;
		}

		double a[10201]; //101x101
		double *a_ptr = a;
		double rptr[101][101];//double **rptr;
		//		a = (double*) rptr;

		double b[101];
		int pivot[101];
		double sol[101];
		memset(rptr,0,sizeof(rptr));
		memset(b,0,sizeof(b));
		memset(pivot,0,sizeof(pivot));
		memset(sol,0,sizeof(sol));

		//----------- Applied Gamma Function for Ordinary Kriging
		for(i=0;i<N;i++){
			for(j=i;j<N;j++){
				if(j==N-1) {
					rptr[i][j] = -1.0;
					//cuPrintf("%f < %f kah?? \n",ptopDistance [i][j],range );
					if (ptopDistance [i][j] < range){
						b[i] = sphericalModel(range, sill, ptopDistance[i][j]);
					}
					else{
						b[i] = sill;
					}
					//					//cuPrintf("b[%d]:%f\n",i,b[i]);
				}else{
					if (ptopDistance [i][j] < range){
						rptr[i][j] = sphericalModel(range, sill, ptopDistance[i][j]);
					}
					else{
						rptr[i][j] = sill;
					}
					//					//cuPrintf("rptr[%d]:%f\n",i,rptr[i]);
					rptr[j][i] = rptr[i][j];
					rptr[N-1][j]=1.0;
				}
			}
		}
		rptr[N-1][N-1] = 0.0;
		b[N-1] = 1.0;

		//TODO change this with more efficient way
		for(i=0;i<N;i++){
			for(j=0;j<N;j++){
				*a_ptr = rptr[i][j];
				a_ptr++;
			}
		}

		int err = Doolittle_LU_Decomposition_with_Pivoting(a, pivot,  N);
		//printf ("after decom\n");
		if (err < 0) {//cuPrintf("matrix is Singular\n");
			//printf(" Matrix A is singular\n");
		}
		else {
			err = Doolittle_LU_with_Pivoting_Solve(a, b, pivot, sol, N);
		}

		weightsum=0.0;
		predict = 0.0;
		//calculate predict value in grid or radius or nearest points
		for (i=0 ; i<N-1 ; i++){
			predict += z_closestNodesIndex[i] * sol[i];
			weightsum += sol[i];
		}

		ptrpredictionResult[dumpsize]  = predict;
		ptr_xgridpoint[dumpsize] = (x_node*gridsizes)+minimumX;
		ptr_ygridpoint[dumpsize] = (y_node*gridsizes)+minimumY;
		dev_debugVar[dumpsize]   = threadID;
	}

}


// main routine that executes on the host
int main(void)
{
	/** Read, parse  input LIDAR data & find min,max value
        and calculate range X & Y
	 **/

	clock_t start_time = clock();
	int lineNumber=0;//1186845;
	char* inputPathLIDARdata = "data/Data4_XYZ_Ground.txt";
	//char* inputPathLIDARdata = "data/DataSample.txt";

	//Calculate number of line from input file
	static const char* filename = inputPathLIDARdata;
	FILE *filetmp = fopen ( filename, "r" );
	if ( filetmp != NULL )
	{
		char line [ 128 ];
		while ( fgets ( line, sizeof line, filetmp ) != NULL ) /* read a line */
		{
			lineNumber++; /* write the line */
		}
		fclose ( filetmp );
	}
	else
		perror ( filename );

	printf("Number of Input Data: %d\n", lineNumber);

	int N = lineNumber;
	FILE *file=NULL;
	int i=0;

	//start Gridding Process
	file=fopen(inputPathLIDARdata,"r");
	if(file==NULL){
		fprintf(stderr,"[gridding.cu] cannot open input LIDAR Data\n");
		exit(EXIT_FAILURE);
	}

	double *x=NULL;
	double *y=NULL;
	double *z=NULL;
	x = (double*)malloc(sizeof(double)*(N));
	y = (double*)malloc(sizeof(double)*(N));
	z = (double*)malloc(sizeof(double)*(N));
	for(i=0;i<N;i++){
		fscanf(file,"%lf %lf %lf", &x[i], &y[i], &z[i]);
		findMinMax(x[i], y[i]);
	}
	fclose (file);

	//round min & max value
	minY = floor(minY);
	minX = floor(minX);
	maxX = ceil(maxX);
	maxY = ceil(maxY);
	printf ("min X %lf; max X %lf\n" , minX, maxX );
	printf ("min Y %lf; max Y %lf\n" , minY, maxY );
	int gridXrange = (int) (maxX-minX );
	int gridYrange = (int) (maxY-minY );
	printf ("gridXrange original data %d (in meters) \n", gridXrange);
	printf ("gridYrange original data %d (in meters) \n", gridYrange);

	/**
	 * Calculate grid size based on desired gridding size
	 */

	double gridsize = 1; // in meter
	int numdevices = 2;  // set with 1 or 2 devices

	//set CUDA thread dimension
	int dimGridsize  = gridXrange/gridsize;
	int dimBlocksize = gridYrange/gridsize;
	int totalGrids = (dimBlocksize)*(dimGridsize);
	int numthreads_pergpu = totalGrids/numdevices;

	printf("Grid size                 : %2.2f m \n",gridsize);
	printf("DimGridsize               : %d\n",dimGridsize);
	printf("DimBlocksize              : %d\n",dimBlocksize);
	printf("Number of TOTAL GPU Grid  : %d threads\n",totalGrids);

	dim3 dimGrid(1024,1);
	dim3 dimBlock(1024,1);

	//Define variable for all CUDA devices
	double *dev_debugVar, *dev_debugVar2; //for debugger purpose
	double *dev_x, *dev_x2, *dev_y, *dev_y2, *dev_z, *dev_z2; //x,y,z store in devices
	double *predictionResult, *predictionResult2; //store prediction result here
	//pointer to result storage in host
	double *host_predictionResult  = (double*) malloc(sizeof(double)*1024*1020);
	double *host_predictionResult2 = (double*) malloc(sizeof(double)*1024*1020);
	memset(host_predictionResult, -1,sizeof(double)*1024*1020);
	memset(host_predictionResult2,-1,sizeof(double)*1024*1020);
	double *dev_y_gridpoint, *dev_x_gridpoint,*dev_y_gridpoint2, *dev_x_gridpoint2;//saving corresponding x,y point in devices
	//saving x,y correponding point in host
	double *x_gridpoint  = (double*) malloc(sizeof(double)*numthreads_pergpu);//device 1
	double *y_gridpoint  = (double*) malloc(sizeof(double)*numthreads_pergpu);//device 1
	double *x_gridpoint2 = (double*) malloc(sizeof(double)*numthreads_pergpu);//device 2
	double *y_gridpoint2 = (double*) malloc(sizeof(double)*numthreads_pergpu);//device 2
	//create debugger storage
	double *host_debugVar  = (double*) malloc(sizeof(double)*numthreads_pergpu);//device 1
	double *host_debugVar2 = (double*) malloc(sizeof(double)*numthreads_pergpu);//device 2
	memset(host_debugVar, -1,sizeof(double)*numthreads_pergpu);
	memset(host_debugVar2,-1,sizeof(double)*numthreads_pergpu);
	int device; //device ID

	int loop=0;
	while(numthreads_pergpu>(1020*1020)){
		numthreads_pergpu = numthreads_pergpu/2;
		loop++;
	}
	printf("numthreads per gpu devices: %d threads/devices\n\n",numthreads_pergpu);
	clock_t preprocessing_time;
	clock_t finished_time;

	preprocessing_time = clock();
	int p;

	printf("Running Kernel threads...\n");
	for(p=0;p<=loop;p++){
		hipSetDevice(0);
		//hipDeviceReset();
		hipGetDevice(&device);
		cudaThreadSetLimit(hipLimitMallocHeapSize,1024*1024*1024);

		hipMalloc((void**)&dev_x,sizeof(double)*(N));
		hipMalloc((void**)&dev_y,sizeof(double)*(N));
		hipMalloc((void**)&dev_z,sizeof(double)*(N));
		hipMalloc((void**)&predictionResult, sizeof(double)*numthreads_pergpu);
		hipMalloc((void**)&dev_x_gridpoint, sizeof(double)*numthreads_pergpu);
		hipMalloc((void**)&dev_y_gridpoint, sizeof(double)*numthreads_pergpu);
		hipMalloc((void**)&dev_debugVar, sizeof(double)*numthreads_pergpu);
		hipMemset((void*)predictionResult,-1,sizeof(double)*numthreads_pergpu);

		hipMemcpy(dev_x, x, sizeof(double)*(N), hipMemcpyHostToDevice);
		hipMemcpy(dev_y, y, sizeof(double)*(N), hipMemcpyHostToDevice);
		hipMemcpy(dev_z, z, sizeof(double)*(N), hipMemcpyHostToDevice);

		//limit threads per device


		on_core_process<<< dimGrid, dimBlock>>>(dev_debugVar,p*numthreads_pergpu,dev_x_gridpoint,dev_y_gridpoint,predictionResult, dev_x, dev_y, dev_z,
				minX,minY, N,gridsize, (p+1)*numthreads_pergpu,dimGridsize);


		printf("[Device %d] CUDA err: %s \n", device,hipGetErrorString(hipGetLastError()));

		//If we need second devices, then start CUDA Programming device 1
		if(numdevices==2){
			hipSetDevice(1);
			//hipDeviceReset();
			hipGetDevice(&device);
			cudaThreadSetLimit(hipLimitMallocHeapSize,1024*1024);

			hipMalloc((void**)&dev_x2,sizeof(double)*(N));
			hipMalloc((void**)&dev_y2,sizeof(double)*(N));
			hipMalloc((void**)&dev_z2,sizeof(double)*(N));
			hipMalloc((void**)&predictionResult2, sizeof(double)*numthreads_pergpu);
			hipMalloc((void**)&dev_x_gridpoint2, sizeof(double)*numthreads_pergpu);
			hipMalloc((void**)&dev_y_gridpoint2, sizeof(double)*numthreads_pergpu);
			hipMalloc((void**)&dev_debugVar2, sizeof(double)*numthreads_pergpu);
			hipMemset((void*)predictionResult2,-1,sizeof(double)*numthreads_pergpu);

			hipMemcpy(dev_x2, x, sizeof(double)*(N), hipMemcpyHostToDevice);
			hipMemcpy(dev_y2, y, sizeof(double)*(N), hipMemcpyHostToDevice);
			hipMemcpy(dev_z2, z, sizeof(double)*(N), hipMemcpyHostToDevice);

			on_core_process<<< dimGrid, dimBlock>>>(dev_debugVar2,(int)((p+1)*numthreads_pergpu),dev_x_gridpoint2,dev_y_gridpoint2,predictionResult2, dev_x2, dev_y2, dev_z2,
					minX,minY, N,gridsize, (p+2)*numthreads_pergpu, dimGridsize);

			printf("[Device %d] CUDA err: %s \n", device,hipGetErrorString(hipGetLastError()));
		}
		//hipDeviceSynchronize();
		//hipDeviceSynchronize();

		hipMemcpy(host_predictionResult,predictionResult,sizeof(double)*numthreads_pergpu,hipMemcpyDeviceToHost);
		hipMemcpy(x_gridpoint,dev_x_gridpoint,sizeof(double)*numthreads_pergpu,hipMemcpyDeviceToHost);
		hipMemcpy(y_gridpoint,dev_y_gridpoint,sizeof(double)*numthreads_pergpu,hipMemcpyDeviceToHost);
		hipMemcpy(host_debugVar,dev_debugVar,sizeof(double)*numthreads_pergpu,hipMemcpyDeviceToHost);


		if(numdevices==2){
			hipMemcpy(host_predictionResult2,predictionResult2,sizeof(double)*numthreads_pergpu,hipMemcpyDeviceToHost);
			hipMemcpy(x_gridpoint2,dev_x_gridpoint2,sizeof(double)*numthreads_pergpu,hipMemcpyDeviceToHost);
			hipMemcpy(y_gridpoint2,dev_y_gridpoint2,sizeof(double)*numthreads_pergpu,hipMemcpyDeviceToHost);
			hipMemcpy(host_debugVar2,dev_debugVar2,sizeof(double)*numthreads_pergpu,hipMemcpyDeviceToHost);
		}

		//lets Write output value to file
		FILE *fout;

		/* open the file */
		fout = fopen("output/prediction_result.txt", "a");
		if (fout == NULL) {
			printf("I couldn't open output/prediction_result.txt for writing.\n");
			exit(0);
		}

		/* write to the file */
		printf("Writting to file ... \n");

		for (i=0; i< numthreads_pergpu; i++){
			fprintf(fout, "%2.2f  ", *(x_gridpoint+i));
			fprintf(fout, "%2.2f  ", *(y_gridpoint+i));
			fprintf(fout, "%2.2f\n", *(host_predictionResult+i));
			if(numdevices==2){
				fprintf(fout, "%2.2f  ", *(x_gridpoint2+i));
				fprintf(fout, "%2.2f  ", *(y_gridpoint2+i));
				fprintf(fout, "%2.2f\n", *(host_predictionResult2+i));
			}
		}
		/* close the file */
		fclose(fout);

		/* create debug file */
		fout = fopen("output/debug.txt", "a");
		if (fout == NULL) {
			printf("I couldn't open output/prediction_result.txt for writing.\n");
			exit(0);
		}

		/* write to the file */
		printf("Writting DEBUG file ... \n");
		fprintf(fout,"threadID written as below (one line one GPU thread): \n");
		for (i=0; i< numthreads_pergpu; i++){
			fprintf(fout, "%2.2f\n", *(host_debugVar+i));
			if(numdevices==2){
				fprintf(fout, "%2.2f\n", *(host_debugVar2+i));
			}
		}
		/* close the file */
		fclose(fout);
		printf("Finished wrote in loop %d\n",p);
	}

	finished_time = clock();
	//	printf("CUDA Synch err: %s \n", hipGetErrorString(hipDeviceSynchronize()));
	printf("CUDA last err: %s \n", hipGetErrorString(hipGetLastError()));

	clock_t postprocessing_time = clock();

	double time_preprocessing 		= ((double)(preprocessing_time-start_time))/CLOCKS_PER_SEC;
	double time_kernelProcessing 	= ((double)(finished_time-preprocessing_time))/CLOCKS_PER_SEC;
	double time_postprocessing	    = ((double)(postprocessing_time-finished_time))/CLOCKS_PER_SEC;

	printf("\ntotal Preprocessing time %2.2f \n",time_preprocessing);
	printf("total Kernel time %2.2f \n",time_kernelProcessing);
	printf("total Postprocessing time %2.2f \n",time_postprocessing);

	//de-allocate memory both in host and devices
	free(x);
	free(y);
	free(z);
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	hipFree(dev_x2);
	hipFree(dev_y2);
	hipFree(dev_z2);
	hipFree(predictionResult);
	hipFree(predictionResult2);
}

void findMinMax (double x, double y){
	if (minX == 0.0 && maxX == 0.0){
		minX = x;
		minY = y;
		maxX = x;
		maxY = y;
	}

	if(minY>y)
		minY = y;
	if(minX>x)
		minX = x;
	if(maxX<x)
		maxX = x;
	if(maxY<y)
		maxY = y;
}